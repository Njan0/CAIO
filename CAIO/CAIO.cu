#include "hip/hip_runtime.h"
#include "CAIO.cuh"

#include <hip/hip_runtime.h>
#include <>

using State = CAIO::State;

std::array<unsigned char, 16> generateStateCount() {
    std::array<unsigned char, 16> result;
    for (int i = 0; i < 16; ++i) {
        // count bits which are one
        int ones = 0;
        for (int bit = 0; bit < 4; ++bit) {
            ones += i >> bit & 1;
        }

        result[i] = ones;
    }

    return result;
}
const std::array<unsigned char, 16> stateCount = generateStateCount();

std::array<unsigned char, 16> generateStateColor() {
    std::array<unsigned char, 16> result;
    for (int i = 0; i < 16; ++i) {
        result[i] = stateCount[i] == 4 ? 255 : stateCount[i] * 64;
    }

    return result;
}
const std::array<unsigned char, 16> stateColor = generateStateColor();

__host__ __device__ State operator&(const State& lhs, const State& rhs)
{
    return static_cast<State>(static_cast<unsigned char>(lhs) & static_cast<unsigned char>(rhs));
}

__host__ __device__ State operator|(const State& lhs, const State& rhs)
{
    return static_cast<State>(static_cast<unsigned char>(lhs) | static_cast<unsigned char>(rhs));
}

__host__ __device__ State& operator|=(State& lhs, const State& rhs)
{
    return lhs = static_cast<State>(static_cast<unsigned char>(lhs) | static_cast<unsigned char>(rhs));
}

CAIO::CAIO(int width, int height, bool reflectiveBoundary, std::array<State, 16> updateRules, const std::function<State(int x, int y)>& states) : width(width), height(height), reflective(reflectiveBoundary), size(static_cast<size_t>(width) * height), cells(), updateRules(updateRules) {
    for (auto rule : updateRules) {
        if (static_cast<unsigned char>(rule) >= 16) {
            throw "Invalid update rule";
        }
    }

    // initalize cells
    cells.reserve(size);
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            auto state = states(x, y);
            cells.emplace_back(state);
            drawBuffer.emplace_back(stateColor[static_cast<char>(state)]);
        }
    }
}

int CAIO::getIndex(int x, int y) {
    return x + y * width;
}

bool CAIO::checkState(int x, int y, State state) {
    return checkState(getIndex(x, y), state);
}

bool CAIO::checkState(int i, State state) {
    return (cells[i] & state) == state;
}

void CAIO::update() {
    auto inputs = std::make_unique<State[]>(size);

    if (reflective) {
        // loop for reflective boundary
        int i = 0;
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                State state = State::Empty;

                if (x > 0) {
                    // check cell to the left
                    if (checkState(x - 1, y, State::Right))
                        state |= State::Left;
                }
                else {
                    // left boundary
                    state |= cells[i] & State::Left;
                }

                if (y > 0) {
                    // check cell above
                    if (checkState(x, y - 1, State::Down))
                        state |= State::Up;
                }
                else {
                    // upper boundary
                    state |= cells[i] & State::Up;
                }

                if (x < width - 1) {
                    // check cell to the right
                    if (checkState(x + 1, y, State::Left))
                        state |= State::Right;
                }
                else {
                    // right boundary
                    state |= cells[i] & State::Right;
                }

                if (y < height - 1) {
                    // check cell below
                    if (checkState(x, y + 1, State::Up))
                        state |= State::Down;
                }
                else {
                    // lower boundary
                    state |= cells[i] & State::Down;
                }

                inputs[i] = state;
                ++i;
            }
        }
    }
    else {
        // loop for periodic boundary
        int i = 0;
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                State state = State::Empty;

                // check cell to the left
                int left = x > 0 ? x - 1 : width - 1;
                if (checkState(left, y, State::Right))
                    state |= State::Left;

                // check cell above
                int up = y > 0 ? y - 1 : height - 1;
                if (checkState(x, up, State::Down))
                    state |= State::Up;

                // check cell to the right
                int right = x < width - 1 ? x + 1 : 0;
                if (checkState(right, y, State::Left))
                    state |= State::Right;

                // check cell below
                int down = y < height - 1 ? y + 1 : 0;
                if (checkState(x, down, State::Up))
                    state |= State::Down;

                inputs[i] = state;
                ++i;
            }
        }
    }

    // apply update rules
    for (int i = 0; i < size; ++i) {
        cells[i] = updateRules[static_cast<unsigned char>(inputs[i])];
        drawBuffer[i] = stateColor[static_cast<char>(cells[i])];
    }
}

void CAIO::draw(std::ostream& out) {
    out.write(drawBuffer.data(), size);
}

__device__ inline int getIndex(int x, int y, int width) {
    return x + y * width;
}

__device__ inline bool checkState(State* cells, int x, int y, int width, State state) {
    return (cells[getIndex(x, y, width)] & state) == state;
}

__global__ void updateKernel(State* cells, int width, int height, bool reflectiveBoundary, State* updateRules, State* out, char* colorRules, char* outColor) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    State input = State::Empty;
    int i = getIndex(x, y, width);

    if (reflectiveBoundary) {
        if (x > 0) {
            // check cell to the left
            if (checkState(cells, x - 1, y, width, State::Right))
                input |= State::Left;
        }
        else {
            // left boundary
            input |= cells[i] & State::Left;
        }

        if (y > 0) {
            // check cell above
            if (checkState(cells, x, y - 1, width, State::Down))
                input |= State::Up;
        }
        else {
            // upper boundary
            input |= cells[i] & State::Up;
        }

        if (x < width - 1) {
            // check cell to the right
            if (checkState(cells, x + 1, y, width, State::Left))
                input |= State::Right;
        }
        else {
            // right boundary
            input |= cells[i] & State::Right;
        }

        if (y < height - 1) {
            // check cell below
            if (checkState(cells, x, y + 1, width, State::Up))
                input |= State::Down;
        }
        else {
            // lower boundary
            input |= cells[i] & State::Down;
        }
    }
    else {
        // check cell to the left
        int left = x > 0 ? x - 1 : width - 1;
        if (checkState(cells, left, y, width, State::Right))
            input |= State::Left;

        // check cell above
        int up = y > 0 ? y - 1 : height - 1;
        if (checkState(cells, x, up, width, State::Down))
            input |= State::Up;

        // check cell to the right
        int right = x < width - 1 ? x + 1 : 0;
        if (checkState(cells, right, y, width, State::Left))
            input |= State::Right;

        // check cell below
        int down = y < height - 1 ? y + 1 : 0;
        if (checkState(cells, x, down, width, State::Up))
            input |= State::Down;
    }

    // apply update rule
    out[i] = updateRules[static_cast<unsigned char>(input)];
    outColor[i] = colorRules[static_cast<unsigned char>(out[i])];
}

/**
 * Calculate the smallest integer which is
 * greater or equal the quotient of the
 * two given positive integers.
 *
 * @param dividend
 * @param divisor
 * @return ceiling(dividend / divisor)
 */
int divideRoundUp(unsigned int dividend, unsigned int divisor) {
    return 1 + ((dividend - 1) / divisor);
}

void CAIO::updateCuda(unsigned int blockWidth, unsigned int blockHeight) {
    dim3 blocks(divideRoundUp(width, blockWidth), divideRoundUp(height, blockHeight));
    dim3 threadsPerBlock(blockWidth, blockHeight);

    // input
    State* cellsDevice;
    State* updateRulesDevice;
    char* colorRulesDevice;

    // output
    State* out;
    char* outColor;

    // memory sizes
    auto dataMemorySize = sizeof(State) * size;
    auto ruleMemorySize = sizeof(State) * 16;
    auto colorMemorySize = sizeof(char) * size;
    auto colorRuleMemorySize = sizeof(char) * 16;

    // copy cells to device
    hipMalloc(&cellsDevice, dataMemorySize);
    hipMemcpy(cellsDevice, cells.data(), dataMemorySize, hipMemcpyHostToDevice);

    // copy rules to device
    hipMalloc(&updateRulesDevice, ruleMemorySize);
    hipMemcpy(updateRulesDevice, updateRules.data(), ruleMemorySize, hipMemcpyHostToDevice);

    // copy color rules to device
    hipMalloc(&colorRulesDevice, colorRuleMemorySize);
    hipMemcpy(colorRulesDevice, stateColor.data(), colorRuleMemorySize, hipMemcpyHostToDevice);

    // allocate memory for result
    hipMalloc(&out, dataMemorySize);
    hipMalloc(&outColor, colorMemorySize);

    // run kernel 
    updateKernel<<<blocks, threadsPerBlock>>>(cellsDevice, width, height, reflective, updateRulesDevice, out, colorRulesDevice, outColor);

    // copy result to host
    hipMemcpy(cells.data(), out, dataMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(drawBuffer.data(), outColor, colorMemorySize, hipMemcpyDeviceToHost);

    // free memory on device
    hipFree(cellsDevice);
    hipFree(updateRulesDevice);
    hipFree(colorRulesDevice);
    hipFree(out);
    hipFree(outColor);
}